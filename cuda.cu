
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define min(x,y) (y + ((x - y) & ((x - y) >> (sizeof(long) * 8 - 1))))

const int Tile_Width = 1;
const int WIDTH = 3;

void print_matrix(long *m) {
	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < WIDTH; j++)
			printf("P[%d][%d] = %ld\n", i, j, m[i * WIDTH + j]);
	printf("\n");
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
	if (result != cudaSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
		assert(result == cudaSuccess);
	}
#endif
	return result;
}


__global__ void reduction(long* Pd, long* Nd, int ndsize) {
	long Pvalue = LONG_MAX;

	int i = blockIdx.y * Tile_Width + threadIdx.y;
	int j = blockIdx.x * Tile_Width + threadIdx.x;

	for (int k = 0; k < ndsize; ++k) {
		Pvalue = min(Pvalue, Nd[k * WIDTH * WIDTH + i * WIDTH + j]);
	}

	Pd[i * WIDTH + j] = Pvalue;
}

void *emalloc(size_t size) {
	void *memory = malloc(size);

	if (!memory) {
		fprintf(stderr, "ERROR: Failed to malloc.\n");
		exit(1);
	}

	return memory;
}


void file_to_matrix(FILE *path_matr, long* m, int ind) {
	/*Reads a file and get the matrix 3x3 from it*/
	// Creating matrix
	char line[20];
	fscanf(path_matr, " %[^\n]", line);
	// Filling matrix
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			fscanf(path_matr, "%ld", &m[(ind * WIDTH * WIDTH) + i * WIDTH + j]);
		}
	}
}


int main(int argc, char* argv[]) {
	if (argc != 2) {
		fprintf(stderr, "Syntax: %s <matrix file>\n", argv[0]);
		return EXIT_FAILURE;
	}

	FILE *path_matr = fopen(argv[1], "r");
	if (path_matr == NULL) {
		fprintf(stderr, "ERROR: Invalid file to matrices.\n");
		exit(1);
	}

	int n_matr; // Number of matrices
	fscanf(path_matr, "%d", &n_matr);


	long* M = (long*) emalloc(n_matr * WIDTH * WIDTH * sizeof(long*));
	long* P = (long*) emalloc(WIDTH * WIDTH * sizeof(long));
 
	for (int i = 0; i < n_matr; i++) {
		file_to_matrix(path_matr, M, i);
	}

	checkCuda( hipSetDevice(0) );
	hipDeviceReset();

	// allocate device matrices (linearized)
	//printf("Allocate device matrices (linearized)...\n");
	long* Nd = NULL;
	long* Pd = NULL;

	checkCuda( hipMalloc((void**) &Nd, n_matr * WIDTH * WIDTH * sizeof(long)) );
	checkCuda( hipMalloc((void**) &Pd, WIDTH * WIDTH * sizeof(long)) );

	struct timeval begin, end;
	gettimeofday(&begin, NULL);
	// copy host memory to device

	// cudaMemcpy -> faz copias de vetores do host para o device
	checkCuda( hipMemcpy(Nd, M, n_matr * WIDTH * WIDTH * sizeof(long), hipMemcpyHostToDevice) );
	checkCuda( hipMemcpy(Pd, P, WIDTH * WIDTH * sizeof(long), hipMemcpyHostToDevice) );
	// execute the kernel
	printf("Execute the kernel...\n");


	int GridSize = (WIDTH + Tile_Width-1) / Tile_Width;
	dim3 gridDim(GridSize, GridSize);
	dim3 blockDim(Tile_Width, Tile_Width);

	hipProfilerStart();
	reduction<<< gridDim, blockDim >>>(Pd, Nd, n_matr);
	hipProfilerStop();

	// copy result from device to host
	checkCuda( hipMemcpy( P, Pd, WIDTH * WIDTH * sizeof(long),hipMemcpyDeviceToHost) );
	gettimeofday(&end, NULL);

	double gpuTime = 1000000*(double)(end.tv_sec - begin.tv_sec);
	gpuTime +=	(double)(end.tv_usec - begin.tv_usec);

	// print times
	printf("Execution Time (microseconds): %9.2f\n", gpuTime);

	print_matrix(P);
	/* print result
	FILE *ptr_file;
	ptr_file =fopen("matMul_gpu_globalmem.out", "w");
	if (!ptr_file) return 1;

	for (int i=0; i < Width; i++){
			for (int j=0; j < Width; j++) fprintf(ptr_file,"%6.2f ", P[i * Width + j]);
			fprintf(ptr_file,"\n");
	}
	fclose(ptr_file); */


	// clean up memory
	free(M);
	free(P);
	checkCuda( hipFree(Nd) );
	checkCuda( hipFree(Pd) );

	return 0;
}